#include <cstdlib>
#include <cstdio>
#include <hip/hip_runtime.h>

#define BLOCK_SIZE 256
#define SLICE_SIZE 784


// A C model derived from the OpenCL kernel 
void softMax_cpu(const int numSlice, const int sliceSize, const float* src, float* dest) {
  for (int i = 0; i < numSlice; i++) {
    float max_ = src[i * sliceSize];
    for (int j = 0; j < sliceSize; j++) {
      max_ = (max_ < src[i * sliceSize + j]) ? src[i * sliceSize + j] : max_;
    }
    float sum = 0;
    for (int j = 0; j < sliceSize; j++) {
      float e = expf(src[i * sliceSize + j] - max_);
      sum += e;
      dest[i * sliceSize + j] = e;
    }
    for (int j = 0; j < sliceSize; j++) {
      dest[i * sliceSize + j] /= sum;
    }
  }
}

__global__ void 
softMax (const int numSlice, const int sliceSize, const float* src, float* dest) {
  unsigned i = blockIdx.x * blockDim.x + threadIdx.x;
  if (i >= numSlice) return;
  float max_ = src[i * sliceSize];
  for (int j = 0; j < sliceSize; j++) {
    max_ = max(max_, src[i * sliceSize + j]);
  }
  float sum = 0;
  for (int j = 0; j < sliceSize; j++) {
    sum += exp(src[i * sliceSize + j] - max_);
  }
  for (int j = 0; j < sliceSize; j++) {
    dest[i * sliceSize + j] = exp(src[i * sliceSize + j] - max_) / sum;
  }
}

int main() {
   
  int numSlice = 10000;
  int sliceSize = SLICE_SIZE;
  int numElem = numSlice * sliceSize;

  float* input = (float*) aligned_alloc(1024, sizeof(float) * numElem);
  float* output_gpu = (float*) aligned_alloc(1024, sizeof(float) * numElem);
  float* output_cpu = (float*) aligned_alloc(1024, sizeof(float) * numElem);

  srand(2);
  for (int i = 0; i < numSlice; i++)
    for (int j = 0; j < sliceSize; j++)
      input[i*sliceSize+j] = rand() % 13; 

  float *d_input, *d_output;
  hipMalloc((void**)&d_input, sizeof(float) * numElem);
  hipMalloc((void**)&d_output, sizeof(float) * numElem);
  hipMemcpy(d_input, input, sizeof(float) * numElem, hipMemcpyHostToDevice);

  dim3 global_work_size ((numSlice+BLOCK_SIZE-1)/BLOCK_SIZE*BLOCK_SIZE);
  dim3 local_work_size (BLOCK_SIZE);

  for (int n = 0; n < 100; n++) {
    softMax<<<global_work_size, local_work_size>>>(numSlice, sliceSize, d_input, d_output);
  }

  hipMemcpy(output_gpu, d_output, sizeof(float) * numElem, hipMemcpyDeviceToHost);

  // verification
  softMax_cpu(numSlice, sliceSize, input, output_cpu);
  for (int i = 0; i < numElem; i++) {
    if (fabsf(output_cpu[i] - output_gpu[i]) > 1e-3) {
      printf("@index %d cpu: %f gpu: %f\n", i, output_cpu[i], output_gpu[i]);
      break;
    }
  }

  free(input);
  free(output_cpu);
  free(output_gpu);
  hipFree(d_input);
  hipFree(d_output);
  return 0;
}

